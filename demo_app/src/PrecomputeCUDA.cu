#include "hip/hip_runtime.h"
// File: PrecomputeCUDA.cpp
// Purpose: Implementation file for precomputed light for CUDA class

#include <iostream>

#include "precomputeCUDA.h"

using namespace nimbus;

/**
Order the spheres from front to back for cumulus
@param org Grid 3D position
@param cen Array of sphere positions
@param numSph Number of spheres
@param dir Sun direction
@param candidates Array of selected candidates
@return Number of candidates
*/

__device__  int orderCumulusCUDA(glm::vec3& org, glm::vec4* pos, int numSph, glm::vec3& dir, glm::vec3* candidates)
{
	int n = 0;
	glm::vec3 aux;

	for (int j = 0; j < numSph; j++)
	{

		glm::vec3 cloudPos = glm::vec3(pos[j]);
		float radius = pos[j].w;

		glm::vec3 temp = org - cloudPos;
		float b = 2.0*glm::dot(dir, temp);
		float c = glm::dot(temp, temp) - radius * radius;

		float disc = b * b - 4.0*c;

		if (disc > 0.0)
		{
			disc = glm::sqrt(disc);
			float t1 = ((-b - disc) / 2.0);
			float t2 = ((-b + disc) / 2.0);

			if (t1 > 0.0 && t2 > 0.0)
			{
				candidates[n] = glm::vec3(t1, t2, j);
				n++;
			}
			else if (t1 <= 0.0  && t2 > 0.0)
			{
				candidates[n] = glm::vec3(0, t2, j);
				n++;
			}
		}
	}

	// Insertion-sort algorithm

	int h;
	for (int i = 1; i < n; i++)
	{
		aux = candidates[i];
		h = i - 1;
		while ((h >= 0) && (aux.x < candidates[h].x))
		{
			candidates[h + 1] = candidates[h];
			h--;
		}
		candidates[h + 1] = aux;

	}


	return n;
}

/**
Order ellipsoids from front to back for 3D mesh based clouds
@param org Grid 3D position
@param cen Array of ellipsoid positions
@param src Source model index
@param dst Final model index
@param dir Sun direction
@param candidates Array of selected candidates
@return Number of candidates
*/

__device__ int orderModelCUDA(glm::vec3& org, glm::mat4* pos, int src, int dst, glm::vec3& dir, glm::vec3* candidates)
{
	int n = 0;
	glm::vec3 aux;

	for (int j = src; j < dst; j++)
	{

		glm::vec3 cloudPos = glm::vec3(pos[j][0]);
		float radius = glm::max(pos[j][1][0], glm::max(pos[j][1][1], pos[j][1][2]));

		glm::vec3 temp = org - cloudPos;
		float b = 2.0*glm::dot(dir, temp);
		float c = glm::dot(temp, temp) - radius * radius;

		float disc = b * b - 4.0*c;

		if (disc > 0.0)
		{
			disc = glm::sqrt(disc);
			float t1 = ((-b - disc) / 2.0);
			float t2 = ((-b + disc) / 2.0);

			if (t1 > 0.0 && t2 > 0.0)
			{
				candidates[n] = glm::vec3(t1, t2, j);
				n++;
			}
			else if (t1 <= 0.0  && t2 > 0.0)
			{
				candidates[n] = glm::vec3(0, t2, j);
				n++;
			}
		}
	}

	// Insertion-sort algorithm

	int h;
	for (int i = 1; i < n; i++)
	{
		aux = candidates[i];
		h = i - 1;
		while ((h >= 0) && (aux.x < candidates[h].x))
		{
			candidates[h + 1] = candidates[h];
			h--;
		}
		candidates[h + 1] = aux;

	}

	return n;
}

/**
The Henyey-Greenstein phase function
@param g Asymmetry factor
@return The phase
*/

__device__ float phaseCUDA(float g)
{

	return 0.0795 * ((1.0 - g * g) / pow(1.0 + g * g - 2.0 *g, 1.5));

}

/**
Trace ray for each cumulus sphere to other spheres in the same cloud and others towards the sun direction
@param org Grid 3D position
@param cen Array of sphere positions
@param numSph Number of spheres
@param dir Sun direction
@return Shadow factor
*/

__device__  float traceRayCumulusCUDA(glm::vec3& org, glm::vec4* cen, int numSph, glm::vec3& dir, float darkLevel)
{
	glm::vec3 candidates[100]; // Allocate space for candidates

	int n = orderCumulusCUDA(org, cen, numSph, dir, candidates);

	if (n == 0) return 1.0f; // No candidates

	float  scatterLight = 0.0;

	float T = 1.0f;

	float tOut;

	float t = candidates[0].x;

	float ph = phaseCUDA(0.3f);

	float totalLight = 0.0f;

	for (int i = 0; i < n; i++)
	{

		if (t > candidates[i].y) // No-duplicate-tracing algorithm
			continue;
		else if (t < candidates[i].x)
			t = candidates[i].x;

		tOut = candidates[i].y;

		glm::vec3 centers = glm::vec3(cen[(int)candidates[i].z]);
		float radius = cen[(int)candidates[i].z].w;

		while (t < tOut) // Iterate spheroid
		{
			glm::vec3  pos = org + t * dir;
			float den = 1.0f - glm::distance(pos, centers) / radius;
			float deltaT = exp(-0.01*den);
			// Scattering                                    
			scatterLight += ph * T* ((darkLevel == 0.2f) ? 0.001f : 0.0001f);
			// Absorted light
			float absortLight = T;
			totalLight = absortLight + scatterLight;
			T *= deltaT;
			if (T < darkLevel)
				return totalLight;
			t += 0.1f;
		}


	}
	return  totalLight;
}

/**
Trace ray for each mesh ellipsoid to other ellipsoids in the same cloud towards the sun direction
@param org Grid 3D position
@param cen Array of ellipsoid positions
@param src Source model index
@param dst Final model index
@param dir Sun direction
@return Shadow factor
*/

__device__ float traceRayModelCUDA(glm::vec3& org, glm::mat4* cen, int src, int dst, glm::vec3& dir, float darkLevel)
{
	glm::vec3 candidates[100]; // Allocate space for candidates

	int n = orderModelCUDA(org, cen, src, dst, dir, candidates);

	if (n == 0) return 1.0f; // No candidates

	float  T = 1.0f, scatterLight = 0.0, totalLight;

	float tIn, tOut;

	tIn = candidates[0].x;
	tOut = candidates[n - 1].y;

	float t = tIn;

	float ph = phaseCUDA(0.9);

	while (t <= tOut) // Iterate thorugh the external ellipsoids
	{
		glm::vec3  pos = org + t * dir;
		float den = 1 - (t - tIn) / (tOut - tIn);
		float deltaT = exp(-0.03*den);

		// Scattering                                    
		scatterLight += ph * T*0.0001;
		// Absroted light
		float absortLight = T;
		totalLight = absortLight + scatterLight;
		T *= deltaT;
		if (T < darkLevel)
			return totalLight;
		t += 0.1;

	}
	return totalLight;
}

/**
Convert grid index to 3D world coordinates
@param index Voxel index
@param vmin Bounding-box min
@param cellSiz Voxel size
@return Grid 3D position
*/

__device__ glm::vec3 indexToCoordCUDA(const glm::vec3& index, const glm::vec3& vmin, const glm::vec3& cellSiz)
{
	return (index + glm::vec3(0.5))*cellSiz + vmin;
}

/**
Iterate all the grid to perform light precomputation in cumulus
@param dev_precomp CUDA 3D grid
@param pos Array of sphere positions
@param numSph Number of ellipsoids
@param sunpos Sun 3D position
@param cellSizX X voxel size
@param cellSizY Y voxel size
@param cellSizZ Z voxel size
@param min Bounding box min coordinates
*/

__global__ void precomputeCumulus(float dev_precomp[][NV][NV], glm::vec4* pos, int numSph, glm::vec3 sunpos, float darkLevel, float cellSizX, float cellSizY, float cellSizZ, glm::vec3 min)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	int k = blockIdx.z*blockDim.z + threadIdx.z;

	if ((i < NV) && (j < NV) && (k < NV)) // Avoid limits ovwerflow
	{
		glm::vec3 coord = indexToCoordCUDA(glm::vec3(i, j, k), min, glm::vec3(cellSizX, cellSizY, cellSizZ));
		glm::vec3 dir = glm::normalize(sunpos - coord);
		dev_precomp[k][j][i] = traceRayCumulusCUDA(coord, pos, numSph, dir, darkLevel);
	}

}

/**
Iterate all the grid to perform light precomputation in 3D meshe
@param dev_precomp CUDA 3D grid
@param pos Array of ellipsoid positions
@param numSph Number of ellipsoids
@param totalSph Total number of ellipsoids
@param bound Index of model for morphing
@param sunpos Sun 3D position
@param cellSizX X voxel size
@param cellSizY Y voxel size
@param cellSizZ Z voxel size
@param min Bounding box min coordinates
*/

__global__ void precomputeModelCUDA(float dev_precomp[][NV][NV], glm::mat4* pos, int numSph, int totalSph, int bound, glm::vec3 sunpos, float darkLevel, float cellSizX, float cellSizY, float cellSizZ, glm::vec3 min)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	int k = blockIdx.z*blockDim.z + threadIdx.z;

	if ((i < NV) && (j < NV) && (k < NV)) // Avoid limits ovwerflow
	{
		glm::vec3 coord = indexToCoordCUDA(glm::vec3(i, j, k), min, glm::vec3(cellSizX, cellSizY, cellSizZ));
		glm::vec3 dir = glm::normalize(sunpos - coord);
		if (bound == 0) // Condition for morphing
			dev_precomp[k][j][i] = traceRayModelCUDA(coord, pos, 0, numSph, dir, darkLevel);
		else dev_precomp[k][j][i] = traceRayModelCUDA(coord, pos, numSph, totalSph, dir, darkLevel);
	}

}

/**
Constructor
*/

PrecomputeCUDA::PrecomputeCUDA()
{

	precomp = nullptr;
	devPrecomp = nullptr;
	devPos = nullptr;

	precomp = (precArray*)malloc(NV*NV*NV * sizeof(float));

	if (!precomp)
		throw nimbus::NimbusException("Can't allocate data in host for CUDA light precomputation", __FILE__, __FUNCTION__, __LINE__);

	hipMalloc((void**)&devPrecomp, NV*NV*NV * sizeof(float));

	if (!devPrecomp)
		throw nimbus::NimbusException("Can't allocate data in device for CUDA light precomputation", __FILE__, __FUNCTION__, __LINE__);


	// Calculate block dimensions in grid

	numBlocksX = (int)ceil((double)(NV / (double)THREADS_X));
	numBlocksY = (int)ceil((double)(NV / (double)THREADS_Y));
	numBlocksZ = (int)ceil((double)(NV / (double)THREADS_Z));

	allocated = false;
}

/**
Set number of spheres/ellipsoids
@param totalSPH Total number of spheres/ellipsoids
*/
void PrecomputeCUDA::setTotalSPH(int totalSPH)
{
	if (allocated)
	{
		hipFree(devPos);
		allocated = false;
	}
		
	if (!allocated)
	{
		std::cout << " LLAMANDO" << std::endl;
#ifdef CUMULUS
		hipMalloc((void**)&devPos, totalSPH * sizeof(glm::vec4));
#else
		hipMalloc((void**)&devPos, totalSPH * sizeof(glm::mat4));
#endif
		this->totalSph = totalSPH;
		allocated = true;
	}
}

/** Retrieve grid size
@return grid size
*/

int PrecomputeCUDA::getVoxelsSize()
{
	return NV;
}

/**
Precompute light for all cumulus clouds
@param pos Array of spheres
@param numSph Number of spheres
@param sundDir Sun direction
@param sunDistance Distance to sun
@param darkLevel Shadow level
@param vmin 3D vector of min bounding box coordinates
@param vmax 3D vector of max bounding box coordinates
@param voxelTextureID OpenGL texture ID
*/

void PrecomputeCUDA::precomputeCloud(glm::vec4* pos, int numSph, int numClouds, glm::vec3& sundir, float sunDistance, float darkLevel, glm::vec3* vmin, glm::vec3* vmax, GLuint* voxelTextureID)
{

	dim3 block(numBlocksX, numBlocksY, numBlocksZ);
	dim3 thread(THREADS_X, THREADS_Y, THREADS_Z);

	glm::vec3 sunpos = sundir * glm::vec3(-sunDistance);

	hipMemcpy(devPos, pos, numSph * sizeof(glm::vec4), hipMemcpyHostToDevice);

	for (int bound = 0; bound < numClouds; bound++)
	{

		glm::vec3  min = vmin[bound]; // Bounding box limits
		glm::vec3  max = vmax[bound];

		float cellSizX = ceil((max.x - min.x)) / NV; // Cell size calculation
		float cellSizY = ceil((max.y - min.y)) / NV;
		float cellSizZ = ceil((max.z - min.z)) / NV;

#ifdef CUMULUS

		precomputeCumulus << <block, thread >> > (devPrecomp, devPos, numSph, sunpos, darkLevel, cellSizX, cellSizY, cellSizZ, min);
#endif
		hipMemcpy(precomp, devPrecomp, NV*NV*NV * sizeof(float), hipMemcpyDeviceToHost);

		// Create OpenGL texture


		glBindTexture(GL_TEXTURE_3D, voxelTextureID[bound]);

		glTexParameterf(GL_TEXTURE_3D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
		glTexParameterf(GL_TEXTURE_3D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
		glTexParameterf(GL_TEXTURE_3D, GL_TEXTURE_WRAP_S, GL_REPEAT);
		glTexParameterf(GL_TEXTURE_3D, GL_TEXTURE_WRAP_T, GL_REPEAT);
		glTexParameterf(GL_TEXTURE_3D, GL_TEXTURE_WRAP_R, GL_REPEAT);

		glTexImage3D(GL_TEXTURE_3D, 0, GL_RED, NV, NV, NV, 0, GL_RED, GL_FLOAT, precomp);
			
	}
	
}


/**
Precompute light for all mesh clouds
@param pos Array of ellipsoids
@param numSph Number of ellipsoids
@param numClouds Number of morphing model
@param sundDir Sun direction
@param sunDistance Distance to sun
@param darkLevel Shadow level
@param vmin 3D vector of min bounding box coordinates
@param vmax 3D vector of max bounding box coordinates
@param voxelTextureID OpenGL texture ID
*/
void PrecomputeCUDA::precomputeModel(glm::mat4* pos, int numSph, int numClouds, glm::vec3& sundir, float sunDistance, float darkLevel, glm::vec3* vmin, glm::vec3* vmax, GLuint* voxelTextureID)
{


	dim3 block(numBlocksX, numBlocksY, numBlocksZ);
	dim3 thread(THREADS_X, THREADS_Y, THREADS_Z);

	glm::vec3 sunpos = sundir * glm::vec3(-sunDistance);

	hipMemcpy(devPos, pos, totalSph * sizeof(glm::mat4), hipMemcpyHostToDevice);

	for (int bound = 0; bound < numClouds; bound++)
	{

		glm::vec3  min = vmin[bound]; // Bounding box limits
		glm::vec3  max = vmax[bound];

		float cellSizX = ceil((max.x - min.x)) / NV; // Cell size calculation
		float cellSizY = ceil((max.y - min.y)) / NV;
		float cellSizZ = ceil((max.z - min.z)) / NV;

#ifdef MODEL
		precomputeModelCUDA << <block, thread >> > (devPrecomp, devPos, numSph, totalSph, bound, sunpos, darkLevel, cellSizX, cellSizY, cellSizZ, min);
#endif
		hipMemcpy(precomp, devPrecomp, NV*NV*NV * sizeof(float), hipMemcpyDeviceToHost);

		// Create OpenGL texture

		glGenTextures(1, &voxelTextureID[bound]);

		glBindTexture(GL_TEXTURE_3D, voxelTextureID[bound]);

		glTexParameterf(GL_TEXTURE_3D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
		glTexParameterf(GL_TEXTURE_3D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
		glTexParameterf(GL_TEXTURE_3D, GL_TEXTURE_WRAP_S, GL_REPEAT);
		glTexParameterf(GL_TEXTURE_3D, GL_TEXTURE_WRAP_T, GL_REPEAT);
		glTexParameterf(GL_TEXTURE_3D, GL_TEXTURE_WRAP_R, GL_REPEAT);

		glTexImage3D(GL_TEXTURE_3D, 0, GL_RED, NV, NV, NV, 0, GL_RED, GL_FLOAT, precomp);
	}
	
}


/**
Destructor
*/
PrecomputeCUDA::~PrecomputeCUDA()
{
	if (allocated)
		hipFree(devPos);
	if (precomp)
		free(precomp);
	if (devPrecomp)
		hipFree(devPrecomp);

}
