#include "hip/hip_runtime.h"
// File: FluidCUDA.cpp
// Purpose: Implementation file for fluid dynamics in CUDA
// Based on the article of Jos Stam

#include "FluidCUDA.h"
#include <iostream>

#define MAX(a,b)            (((a) > (b)) ? (a) : (b))
#define LINEARSOLVERTIMES 10
#define SWAP(x0,x) {nRarray *tmp=x0;x0=x;x=tmp;} // Exchange pointers

using namespace nimbus;

/**
Constructor
@param dt Time spacing between the snapshots
@param visc Fluid viscosity
@param diff Diffusion rate
*/
FluidCUDA::FluidCUDA(float dt, float diff, float visc)
{
	this->dt = dt;
	this->diff = diff;
	this->visc = visc;

	allocateData();

	clearData();

	numBlocksX = (int)ceil((float)(M + 2) / (float)THREADS_X);
	numBlocksY = (int)ceil((float)(N + 2) / (float)THREADS_Y);
	numBlocksZ = (int)ceil((float)(O + 2) / (float)THREADS_Z);

	clearUVW();
}


/**
Set wind U component force (X,Y,Z)
@param force Wind force
@param i-index
@param j-index
@param k-index
*/
void FluidCUDA::setUForce(float force, int i, int j, int k)
{
	u[i][j][k] = force;
}

/**
Set wind V component force (X,Y,Z)
@param force Wind force
@param i-index
@param j-index
@param k-index
*/
void FluidCUDA::setVForce(float force, int i, int j, int k)
{
	v[i][j][k] = force;
}

/**
Set wind W component force (X,Y,Z)
@param force Wind force
@param i-index
@param j-index
@param k-index
*/
void FluidCUDA::setWForce(float force, int i, int j, int k)
{
	w[i][j][k] = force;
}

/**
Retrieve wind U component force (X,Y,Z)
@param i-index
@param j-index
@param k-index
@return U force
*/

float FluidCUDA::getUForce(int i, int j, int k)
{
	return u[i][j][k];
}

/**
Retrieve wind V component force (X,Y,Z)
@param i-index
@param j-index
@param k-index
@return V force
*/
float FluidCUDA::getVForce(int i, int j, int k)
{
	return v[i][j][k];
}

/**
Retrieve wind W component force (X,Y,Z)
@param i-index
@param j-index
@param k-index
@return W force
*/

float FluidCUDA::getWForce(int i, int j, int k)
{
	return w[i][j][k];
}

/**
Retrieve grid dimensions
@param 3D grid dimensions
*/
glm::ivec3 FluidCUDA::getDimensions()
{
	return glm::ivec3(M, N, O);
}

/**
Clear FluidCUDA data
*/
void FluidCUDA::clearData()
{

	for (int i = 0; i < M + 2; i++)
		for (int j = 0; j < N + 2; j++)
			for (int k = 0; k < O + 2; k++)
				u[i][j][k] = v[i][j][k] = w[i][j][k] = 0.0;
}

/**
Clear pre-data in device
*/
__global__ void clearUVWDev(float x[][N + 2][O + 2], float y[][N + 2][O + 2], float z[][N + 2][O + 2])
{


	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	int k = blockIdx.z*blockDim.z + threadIdx.z;

	if ((i >= 1) && (j >= 1) && (k >= 1) && (i <= M) && (j <= N) && (k <= O))
		x[i][j][k] = y[i][j][k] = z[i][j][k] = 0.0;
}

/**
Clear pre-data
*/
void FluidCUDA::clearUVW()
{
	dim3 block(numBlocksX, numBlocksY, numBlocksZ);
	dim3 thread(THREADS_X, THREADS_Y, THREADS_Z);
	clearUVWDev << <block, thread >> > (devUPrev, devVPrev, devWPrev);
}

/**
Allocate Fluid CUDA data
*/
int FluidCUDA::allocateData()
{
	size = (M + 2)*(N + 2)*(O + 2);

	u = (nRarray*)malloc(size * sizeof(float));
	v = (nRarray*)malloc(size * sizeof(float));
	w = (nRarray*)malloc(size * sizeof(float));

	if (!u || !v || !w)
		throw nimbus::NimbusException("Can't allocate data in host for CUDA fluids", __FILE__, __FUNCTION__, __LINE__);


	hipMalloc((void**)&devU, size * sizeof(float));
	hipMalloc((void**)&devV, size * sizeof(float));
	hipMalloc((void**)&devW, size * sizeof(float));
	hipMalloc((void**)&devUPrev, size * sizeof(float));
	hipMalloc((void**)&devVPrev, size * sizeof(float));
	hipMalloc((void**)&devWPrev, size * sizeof(float));

	hipMalloc((void**)&devUy, size * sizeof(float));
	hipMalloc((void**)&devVy, size * sizeof(float));
	hipMalloc((void**)&devWy, size * sizeof(float));
	hipMalloc((void**)&devUPrevy, size * sizeof(float));


	if (!devU || !devV || !devW || !devUPrev || !devVPrev || !devWPrev || !devUy || !devVy || !devWy || !devUPrevy)
		throw nimbus::NimbusException("Can't allocate data in device for CUDA fluids", __FILE__, __FUNCTION__, __LINE__);

	return 1;
}

/**
Free FluidCUDA data
*/
void FluidCUDA::freeData()
{
	if (u) free(u);
	if (v) free(v);
	if (w) free(w);

	if (devU) hipFree(devU);
	if (devV) hipFree(devV);
	if (devW) hipFree(devW);
	if (devUPrev) hipFree(devUPrev);
	if (devVPrev) hipFree(devVPrev);
	if (devWPrev) hipFree(devWPrev);

	if (devUy) hipFree(devUy);
	if (devVy) hipFree(devVy);
	if (devWy) hipFree(devWy);
	if (devUPrevy) hipFree(devUPrevy);

}

/**
Send data to device
*/
void FluidCUDA::sendData()
{
	hipMemcpy(devU, u, size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(devV, v, size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(devW, w, size * sizeof(float), hipMemcpyHostToDevice);
}

/**
Received calculated data
*/
void FluidCUDA::receiveData()
{
	hipMemcpy(u, devU, size * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(v, devV, size * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(w, devW, size * sizeof(float), hipMemcpyDeviceToHost);
}

/**
Simulate FluidCUDA
*/
void FluidCUDA::sim()
{
	velStep();
}


/**
Add source
*/

__global__ void addSource(float  x[][N + 2][O + 2], float s[][N + 2][O + 2], float dt)
{

	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	int k = blockIdx.z*blockDim.z + threadIdx.z;

	if ((i >= 1) && (j >= 1) && (k >= 1) && (i <= M) && (j <= N) && (k <= O))
		x[i][j][k] += dt * s[i][j][k];
}


/**
Linear solve
*/
__global__ void kernelLinSolve(float  x[][N + 2][O + 2], float x0[][N + 2][O + 2], float y[][N + 2][O + 2], float a, float c)
{


	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	int k = blockIdx.z*blockDim.z + threadIdx.z;


	if ((i >= 1) && (j >= 1) && (k >= 1) && (i <= M) && (j <= N) && (k <= O))
	{
		// update for each cell
		y[i][j][k] = (x0[i][j][k] + a * (x[i - 1][j][k] + x[i + 1][j][k] + x[i][j - 1][k] + x[i][j + 1][k] + x[i][j][k - 1] + x[i][j][k + 1])) / c;
	}

}

/**
Diffuse
*/
void FluidCUDA::diffuse(float x[][N + 2][O + 2], float x0[][N + 2][O + 2], float diff, float dt, float dev_y[][N + 2][O + 2])
{
	int max = MAX(MAX(M, N), MAX(N, O));
	float a = dt * diff*max*max*max;

	dim3 block(numBlocksX, numBlocksY, numBlocksZ);
	dim3 thread(THREADS_X, THREADS_Y, THREADS_Z);

	for (int k = 0; k < LINEARSOLVERTIMES; k++)
	{
		kernelLinSolve << <block, thread >> > (x, x0, dev_y, a, 1.0f + 6.0f * a);
		SWAP(x, dev_y);
	}

}

/**
Advect
*/

__global__ void kernelAdvect(float d[][N + 2][O + 2], float d0[][N + 2][O + 2], float u[][N + 2][O + 2], float v[][N + 2][O + 2], float w[][N + 2][O + 2], float dt, float dtx, float dty, float dtz)
{


	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	int k = blockIdx.z*blockDim.z + threadIdx.z;

	int i0, j0, k0, i1, j1, k1;
	float x, y, z, s0, t0, s1, t1, u1, u0;



	if ((i >= 1) && (j >= 1) && (k >= 1) && (i <= M) && (j <= N) && (k <= O))
	{

		x = i - dtx * u[i][j][k]; y = j - dty * v[i][j][k]; z = k - dtz * w[i][j][k];
		if (x < 0.5) x = 0.5; if (x > M + 0.5) x = M + 0.5; i0 = (int)x; i1 = i0 + 1;
		if (y < 0.5) y = 0.5; if (y > N + 0.5) y = N + 0.5; j0 = (int)y; j1 = j0 + 1;
		if (z < 0.5) z = 0.5; if (z > O + 0.5) z = O + 0.5; k0 = (int)z; k1 = k0 + 1;

		s1 = x - i0; s0 = 1 - s1; t1 = y - j0; t0 = 1 - t1; u1 = z - k0; u0 = 1 - u1;
		d[i][j][k] = s0 * (t0*u0*d0[i0][j0][k0] + t1 * u0*d0[i0][j1][k0] + t0 * u1*d0[i0][j0][k1] + t1 * u1*d0[i0][j1][k1]) +
			s1 * (t0*u0*d0[i1][j0][k0] + t1 * u0*d0[i1][j1][k0] + t0 * u1*d0[i1][j0][k1] + t1 * u1*d0[i1][j1][k1]);
	}



}

/**
Basic idea behind advection step: look for particles which end up exactly at the cell centers by tracking backwards in time from the cell centers (with a linear backtrace)
*/

void FluidCUDA::advect(float d[][N + 2][O + 2], float d0[][N + 2][O + 2], float u[][N + 2][O + 2], float v[][N + 2][O + 2], float w[][N + 2][O + 2], float dt)
{


	dim3 block(numBlocksX, numBlocksY, numBlocksZ);
	dim3 thread(THREADS_X, THREADS_Y, THREADS_Z);

	float dtx, dty, dtz;

	dtx = dty = dtz = dt * MAX(MAX(M, N), MAX(N, O));

	kernelAdvect << < block, thread >> > (d, d0, u, v, w, dt, dtx, dty, dtz);


}

/**
Project part 1
*/

__global__ void kernelProject1(float u[][N + 2][O + 2], float v[][N + 2][O + 2], float w[][N + 2][O + 2], float p[][N + 2][O + 2], float div[][N + 2][O + 2])
{
	int i = blockIdx.x*blockDim.x + threadIdx.x + 1;
	int j = blockIdx.y*blockDim.y + threadIdx.y + 1;
	int k = blockIdx.z*blockDim.z + threadIdx.z + 1;


	if ((i >= 1) && (j >= 1) && (k >= 1) && (i <= M) && (j <= N) && (k <= O))
	{
		div[i][j][k] = -1.0 / 3.0*((u[i + 1][j][k] - u[i - 1][j][k]) / M + (v[i][j + 1][k] - v[i][j - 1][k]) / M + (w[i][j][k + 1] - w[i][j][k - 1]) / M);
		p[i][j][k] = 0.0;
	}
}

/**
Project part 2
*/
__global__ void kernelProject2(float u[][N + 2][O + 2], float v[][N + 2][O + 2], float w[][N + 2][O + 2], float p[][N + 2][O + 2], float div[][N + 2][O + 2])
{
	int i = blockIdx.x*blockDim.x + threadIdx.x + 1;
	int j = blockIdx.y*blockDim.y + threadIdx.y + 1;
	int k = blockIdx.z*blockDim.z + threadIdx.z + 1;



	if ((i >= 1) && (j >= 1) && (k >= 1) && (i <= M) && (j <= N) && (k <= O))
	{

		u[i][j][k] -= 0.5*M*(p[i + 1][j][k] - p[i - 1][j][k]);
		v[i][j][k] -= 0.5*M*(p[i][j + 1][k] - p[i][j - 1][k]);
		w[i][j][k] -= 0.5*M*(p[i][j][k + 1] - p[i][j][k - 1]);
	}

}


/**
Computing the height field involves the solution of some linear system called a Poisson equation - reuse Gauss-Seidel relaxation code from the density diffusion function.
*/
void FluidCUDA::project(float u[][N + 2][O + 2], float v[][N + 2][O + 2], float w[][N + 2][O + 2], float p[][N + 2][O + 2], float div[][N + 2][O + 2], float dev_y[][N + 2][O + 2])
{

	dim3 block(numBlocksX, numBlocksY, numBlocksZ);
	dim3 thread(THREADS_X, THREADS_Y, THREADS_Z);

	kernelProject1 << < block, thread >> > (u, v, w, p, div);

	nRarray* aux = p;

	for (int k = 0; k < LINEARSOLVERTIMES; k++)
	{
		kernelLinSolve << < block, thread >> > (p, div, dev_y, 1, 6); SWAP(p, dev_y);
	}

	p = aux;

	kernelProject2 << < block, thread >> > (u, v, w, p, div);

}

/**
Simulation step
*/
void FluidCUDA::velStep()
{
	dim3 block(numBlocksX, numBlocksY, numBlocksZ);
	dim3 thread(THREADS_X, THREADS_Y, THREADS_Z);

	addSource << <block, thread >> > (devU, devUPrev, dt); addSource << <block, thread >> > (devV, devVPrev, dt); addSource << <block, thread >> > (devW, devWPrev, dt);
	SWAP(devUPrev, devU);
	diffuse(devU, devUPrev, visc, dt, devUy);
	SWAP(devVPrev, devV);
	diffuse(devV, devVPrev, visc, dt, devVy);
	SWAP(devWPrev, devW);
	diffuse(devW, devWPrev, visc, dt, devWy);
	project(devU, devV, devW, devUPrev, devVPrev, devUPrevy);
	SWAP(devUPrev, devU); SWAP(devVPrev, devV); SWAP(devWPrev, devW);
	advect(devU, devUPrev, devUPrev, devVPrev, devWPrev, dt); advect(devV, devVPrev, devUPrev, devVPrev, devWPrev, dt); advect(devW, devWPrev, devUPrev, devVPrev, devWPrev, dt);
	project(devU, devV, devW, devUPrev, devVPrev, devUPrevy);
}

/**
Destructor
*/
FluidCUDA::~FluidCUDA()
{
	freeData();
}

